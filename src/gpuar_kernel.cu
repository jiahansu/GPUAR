#include "hip/hip_runtime.h"
/***************************************************************************
*                 Arithmetic Encoding and Decoding Library
*
*   Purpose : Use arithmetic coding to compress/decompress streams
*   Original author for host code implementation: Michael Dipperstein
*   Modify to CUDA implementation: Jia-Han Su.
*   Date    : September 2, 2009
*
*****************************************************************************/
#include "gpuar.h"
#include "assert.h"

//texture<unsigned char, 1, hipReadModeElementType> tex;

__device__ __constant__ AdaptiveProbabilityRange INITIALIZED_RANGE[1];
__device__ __constant__ probability_t INITIALIZED_CUMULATIVE_PROB;

__host__ __device__ unsigned int read(void const *src, unsigned int bytes)
{

	unsigned char *p = (unsigned char*)src;
	switch (bytes)
	{
		case 4:
			return(*p | *(p + 1) << 8 | *(p + 2) << 16 | *(p + 3) << 24);
		case 3: 
			return(*p | *(p + 1) << 8 | *(p + 2) << 16);
		case 2:
			return(*p | *(p + 1) << 8);
		case 1: 
			return(*p);
	}
	return 0;
}

__host__ __device__ unsigned short getCompressedSize(const void* src){

	return read(((const char*)src),2);
}

__host__ __device__ unsigned short getUncompressedSize(const void* src){

	return read(((const char*)src)+2,2);
}

__host__ __device__ void write(unsigned int f, void *dst, size_t bytes)
{
	
	unsigned char *p = (unsigned char*)dst;

	switch (bytes)
	{
		case 4: 
			*p = (unsigned char)f;
			*(p + 1) = (unsigned char)(f >> 8);
			*(p + 2) = (unsigned char)(f >> 16);
			*(p + 3) = (unsigned char)(f >> 24);
			return;
		case 3:
			*p = (unsigned char)f;
			*(p + 1) = (unsigned char)(f >> 8);
			*(p + 2) = (unsigned char)(f >> 16);
			return;
		case 2:
			*p = (unsigned char)f;
			*(p + 1) = (unsigned char)(f >> 8);
			return;
		case 1:
			*p = (unsigned char)f;
			return;
	}
}




__host__ __device__ __inline int putChar(const int c, BitPointer* stream){
	stream->fp[0] = c;
	++stream->fp;


//		std::cout<<c<<std::endl;

	return c;
}

/***************************************************************************
*   Function   : writeChar
*   Description: This function writes the byte passed as a parameter to the
*                stream passed a parameter.
*   Parameters : c - the character to be written
*                stream - pointer to bit stream to write to
*   Effects    : Writes a byte to the file and updates buffer accordingly.
*   Returned   : On success, the character written, otherwise EOF.
***************************************************************************/
__host__ __device__ __inline int writeChar(const int c, BitPointer *stream)
{
    unsigned char tmp;

    if (stream->bitCount == 0)
    {
        /* we can just put byte from file */
		return putChar(c,stream);
        //return fputc(c, stream->fp);
    }else{

		/* figure out what to write */
		tmp = ((unsigned char)c) >> (stream->bitCount);
		tmp = tmp | ((stream->bitBuffer) << (8 - stream->bitCount));
		putChar(tmp, stream);
		stream->bitBuffer = c;


		return tmp;
    }
}

/***************************************************************************
*   Function   : writeBit
*   Description: This function writes the bit passed as a parameter to the
*                file passed a parameter.
*   Parameters : c - the bit value to be written
*                stream - pointer to bit  stream to write to
*   Effects    : Writes a bit to the bit buffer.  If the buffer has a byte,
*                the buffer is written to the file and cleared.
*   Returned   : On success, the bit value written, otherwise EOF.
***************************************************************************/
__host__ __device__ __inline int writeBit(const int c, BitPointer *stream)
{
    int returnValue = c;

    stream->bitCount++;
    stream->bitBuffer <<= 1;

    if (c != 0)
    {
        stream->bitBuffer |= 1;
    }

    /* write bit buffer if we have 8 bits */
    if (stream->bitCount == 8)
    {
		putChar(stream->bitBuffer,stream);

        /* reset buffer */
        stream->bitCount = 0;
        stream->bitBuffer = 0;
    }

    return returnValue;
}

/***************************************************************************
*   Function   : writeBits   (Little Endian)
*   Description: This function writes the specified number of bits from the
*                memory location passed as a parameter to the file passed
*                as a parameter.   Bits are written LSB to MSB.
*   Parameters : stream - pointer to bit stream to write to
*                bits - pointer to bits to write
*                count - number of bits to write
*   Effects    : Writes bits to the bit buffer and file stream.  The bit
*                buffer will be modified as necessary.  bits is treated as
*                a little endian integer of length >= (count/8) + 1.
*   Returned   : EOF for failure, otherwise the number of bits written.  If
*                an error occurs after a partial write, the partially
*                written bits will not be unwritten.
***************************************************************************/
__host__ __device__ __inline int writeBits(BitPointer *stream, void *bits, const unsigned int count)
{
    unsigned char *bytes, tmp;
    int offset, remaining/*, returnValue*/;

    bytes = (unsigned char *)bits;
    offset = 0;
    remaining = count;

    /* write whole bytes */
    while (remaining >= 8)
    {
		writeChar(bytes[offset],stream);
		//returnValue = BitFilePutChar(bytes[offset], stream);

        remaining -= 8;
        offset++;
    }

    if (remaining != 0)
    {
        /* write remaining bits */
        tmp = bytes[offset];
        tmp <<= (8 - remaining);

        while (remaining > 0)
        {
            /*returnValue =*/ writeBit((tmp & 0x80), stream);

            tmp <<= 1;
            remaining--;
        }
    }

    return count;
}


__host__ __device__ __inline int forward(const int symbol){
	return symbol+(symbol&(-symbol));
}

__host__ __device__ __inline int backward(const int symbol){
	return symbol&(symbol-1);
}

__host__ __device__ __inline probability_t getRange(const int symbol,  AdaptiveProbabilityRange& r){
	int i=symbol;
	probability_t h=0;

	while(i!=0){
		h = h+r.ranges[i];
		i = backward(i);
	}

	return h;
}

__host__ __device__ __inline void update(const int symbol,  AdaptiveProbabilityRange& r){
	int i=symbol;

	while(i<=UPPER(EOF_CHAR)){
		r.ranges[i] = r.ranges[i]+1;
		i = forward(i);
	}

}

/***************************************************************************
*   Function   : applySymbolRange
*   Description: This function is used for both encoding and decoding.  It
*                applies the range restrictions of a new symbol to the
*                current upper and lower range bounds of an encoded stream.
*                If an adaptive model is being used, the probability range
*                list will be updated after the effect of the symbol is
*                applied.
*   Parameters : symbol - The symbol to be added to the current code range
*
*   Effects    : The current upper and lower range bounds are adjusted to
*                include the range effects of adding another symbol to the
*                encoded stream.  If an adaptive model is being used, the
*                probability range list will be updated.
*   Returned   : None
***************************************************************************/
__host__ __device__ __inline void applySymbolRange(const int symbol,  AdaptiveProbabilityRange& r, probability_t &lower,probability_t &upper, probability_t &cumulativeProb)
{
    unsigned int range;        /* must be able to hold max upper + 1 */
    unsigned int rescaled;     /* range rescaled for range of new symbol */
                                /* must hold range * max upper */

    /* for updating dynamic models */
    //int i;
    

    /***********************************************************************
    * Calculate new upper and lower ranges.  Since the new upper range is
    * dependant of the old lower range, compute the upper range first.
    ***********************************************************************/
    range = (unsigned int)(upper - lower) + 1;         /* current range */

    /* scale upper range of the symbol being coded */
    rescaled = (unsigned int)getRange(UPPER(symbol),r) * range;
    rescaled /= (unsigned int)cumulativeProb;

    /* new upper = old lower + rescaled new upper - 1*/
    upper = lower + (probability_t)rescaled - 1;

    /* scale lower range of the symbol being coded */
    rescaled = (unsigned int)getRange(LOWER(symbol),r) * range;
    rescaled /= (unsigned int)cumulativeProb;

    /* new lower = old lower + rescaled new upper */
    lower = lower + (probability_t)rescaled;


        /* add new symbol to model */
        ++cumulativeProb;
		
        update(UPPER(symbol),r);
		/*
		for (i = UPPER(symbol); i <= UPPER(EOF_CHAR); i++)
        {
            
            r.ranges[i] += 1;
        }*/

        /* half current values if cumulativeProb is too large */
		  /*
        if (r.cumulativeProb >= MAX_PROBABILITY)
        {
            probability_t original=0;    
			probability_t delta;       

			r.cumulativeProb = 0;
            
			#pragma unroll 256
            for (i = 1; i <= UPPER(EOF_CHAR); i++)
            {
                delta = r.ranges[i] - original;
                if (delta <= 2)
                {
                    
                    original = r.ranges[i];
                    r.ranges[i] = r.ranges[i - 1] + 1;
                }
                else
                {
                    original = r.ranges[i];
                    r.ranges[i] = r.ranges[i - 1] + (delta >>1);
                }

                r.cumulativeProb += (r.ranges[i] - r.ranges[i - 1]);
            }
        }*/


#ifdef _DEBUG
    if (lower > upper)
    {
        /* compile this in when testing new models. */
        assert("Panic: out of range");
		//std::cout<< "Panic: lower ("<< lower<<")> upper ("<<upper<<std::endl;
        //fprintf(stderr, "Panic: lower (%X)> upper (%X)\n", lower, upper);
    }
#endif
}

/***************************************************************************
*   Function   : writeEncodedBits
*   Description: This function attempts to shift out as many code bits as
*                possible, writing the shifted bits to the encoded output
*                file.  Only bits that will be unchanged when additional
*                symbols are encoded may be written out.
*
*                If the n most significant bits of the lower and upper range
*                bounds match, they will not be changed when additional
*                symbols are encoded, so they may be shifted out.
*
*                Adjustments are also made to prevent possible underflows
*                that occur when the upper and lower ranges are so close
*                that encoding another symbol won't change their values.
*   Parameters : bfpOut - pointer to open stream to write to.
*   Effects    : The upper and lower code bounds are adjusted so that they
*                only contain only bits that may be affected by the
*                addition of a new symbol to the encoded stream.
*   Returned   : None
***************************************************************************/
__host__ __device__ __inline void writeEncodedBits(BitPointer *bfpOut, probability_t &lower,probability_t &upper, probability_t &underflowBits)
{
    for (;;)
    {
        if ((upper & MASK_BIT(0)) == (lower & MASK_BIT(0)))
        {
            /* MSBs match, write them to output file */
            writeBit((upper & MASK_BIT(0)) != 0, bfpOut);

            /* we can write out underflow bits too */
            while (underflowBits > 0)
            {
                writeBit((upper & MASK_BIT(0)) == 0, bfpOut);
                underflowBits--;
            }
        }
        else if ((lower & MASK_BIT(1)) && !(upper & MASK_BIT(1)))
        {
            /***************************************************************
            * Possible underflow condition: neither MSBs nor second MSBs
            * match.  It must be the case that lower and upper have MSBs of
            * 01 and 10.  Remove 2nd MSB from lower and upper.
            ***************************************************************/
            underflowBits += 1;
            lower &= ~(MASK_BIT(0) | MASK_BIT(1));
            upper |= MASK_BIT(1);

            /***************************************************************
            * The shifts below make the rest of the bit removal work.  If
            * you don't believe me try it yourself.
            ***************************************************************/
        }
        else
        {
            /* nothing left to do */
            return ;
        }

        /*******************************************************************
        * Shift out old MSB and shift in new LSB.  Remember that lower has
        * all 0s beyond it's end and upper has all 1s beyond it's end.
        *******************************************************************/
        lower <<= 1;
        upper <<= 1;
        upper |= 1;
    }
}

/***************************************************************************
*   Function   : writeRemaining
*   Description: This function writes out all remaining significant bits
*                in the upper and lower ranges and the underflow bits once
*                the last symbol has been encoded.
*   Parameters : bfpOut - pointer to open stream to write to.
*   Effects    : Remaining significant range bits are written to the output
*                file.
*   Returned   : None
***************************************************************************/
__host__ __device__ __inline void writeRemaining(BitPointer *bfpOut,probability_t &lower,probability_t &upper,probability_t &underflowBits      /* current underflow bit count */)
{
    writeBit((lower & MASK_BIT(1)) != 0, bfpOut);

    /* write out any unwritten underflow bits */
    for (underflowBits++; underflowBits > 0; underflowBits--)
    {
        writeBit((lower & MASK_BIT(1)) == 0, bfpOut);
    }
}

/***************************************************************************
*   Function   : initializeAdaptiveProbabilityRangeList
*   Description: This routine builds the initial global list of upper and
*                lower probability ranges for each symbol.  This routine
*                is used by both adaptive encoding and decoding.
*                Currently it provides a uniform symbol distribution.
*                Other distributions might be better suited for known data
*                types (such as English text).
*   Parameters : NONE
*   Effects    : ranges array is made to contain initial probability ranges
*                for each symbol.
*   Returned   : NONE
***************************************************************************/
__host__ __device__ __inline void initializeAdaptiveProbabilityRangeList(AdaptiveProbabilityRange* r, probability_t& cumulativeProb)
{
    int c;

    cumulativeProb = 0;
    r->ranges[0] = 0;          /* absolute lower range */
	
	
    /* assign upper and lower probability ranges assuming */
	memset(r->ranges,0,sizeof(probability_t)*(UPPER(EOF_CHAR)+1) );
    #pragma unroll 256
    for (c = 1; c <= UPPER(EOF_CHAR); c++)
    {
        
		update(c,*r);
        ++cumulativeProb;
    }
}

/***************************************************************************
*   Function   : writeClose
*   Description: This function closes a bit file and frees all associated
*                data.
*   Parameters : stream - pointer to bit stream being closed
*   Effects    : The specified file will be closed and the file structure
*                will be freed.
*   Returned   : 0 for success or EOF for failure.
***************************************************************************/
__host__ __device__ __inline void writeClose(BitPointer *stream)
{

    /* write out any unwritten bits */
    if (stream->bitCount != 0)
    {
        (stream->bitBuffer) <<= 8 - (stream->bitCount);
        putChar(stream->bitBuffer, stream);   /* handle error? */
     }
}


__host__ __device__ BitPointer createBitPointer(unsigned char *data){
	BitPointer bp;
	
	bp.fp = data;                 /* file pointer used by stdio functions */
                  
    bp.bitBuffer=0;    /* bits waiting to be read/written */
    bp.bitCount=0;     /* number of bits in bitBuffer */
    
    return bp;
}	



__host__ void initConstantRange(){
	AdaptiveProbabilityRange r;
	probability_t cumulativeProb;
	initializeAdaptiveProbabilityRangeList(&r,cumulativeProb);
	hipMemcpyToSymbol(HIP_SYMBOL(INITIALIZED_RANGE[0]),&r,sizeof(AdaptiveProbabilityRange));
	hipMemcpyToSymbol(HIP_SYMBOL(INITIALIZED_CUMULATIVE_PROB), &cumulativeProb,sizeof(probability_t));
}

/***************************************************************************
*   Function   : arCompress
*   Description: This routine generates a list of arithmetic code ranges for
*                a file and then uses them to write out an encoded version
*                of that file.
*   Parameters : inFile - Pointer of stream to encode
*                outFile - Pointer of stream to write encoded output to
*   Effects    : Binary data is arithmetically encoded
*   Returned   : TRUE for success, otherwise FALSE.
***************************************************************************/
__host__ __device__ size_t arCompress(const unsigned char *fpIn,const size_t size ,unsigned char *outFile,AdaptiveProbabilityRange& r, probability_t& cumulativeProb)
{
     unsigned char c;
    BitPointer bfpOut = createBitPointer(outFile+PACKET_HEADER_LENGTH);                 /* encoded output */
	
	    /* initialize coder start with full probability range [0%, 100%) */
    probability_t lower = 0;
    probability_t upper = ~0;                     /* all ones */
    probability_t underflowBits = 0;
	size_t length;
	ulonglong2 element;
	ulonglong2* elementPointer = (ulonglong2*)fpIn;
	size_t elementCount = ceil((float)size/(float)sizeof(ulonglong2));
	size_t remaining=size;
	unsigned int bytesOffset;

	

     /* initialize probability ranges asumming uniform distribution */
    /*
	#ifdef _DEBUG
	if(r->cumulativeProb!=256){
		assert("AdaptiveProbabilityRange was not initialized yet!");
	}

	#endif
	*/
	for(size_t i=0;i<elementCount;++i){
		element = elementPointer[i];
		bytesOffset = 0;
		//dataPointer = (unsigned char*)(&element);
		/* encode symbols one at a time */
		
		while(bytesOffset<sizeof(unsigned long long)&&remaining>0)
		{
			c =(unsigned char)(element.x >> (bytesOffset*8) );
			//c = fpIn[j];
			applySymbolRange(c,r,lower,upper, cumulativeProb);
			writeEncodedBits(&bfpOut,lower,upper,underflowBits);
			++bytesOffset;
			--remaining;
		}

		bytesOffset = 0;
		while(bytesOffset<sizeof(unsigned long long)&&remaining>0)
		{
			c =(unsigned char)(element.y >> (bytesOffset*8) );
			//c = fpIn[j];
			applySymbolRange(c,r,lower,upper,cumulativeProb);
			writeEncodedBits(&bfpOut,lower,upper,underflowBits);
			++bytesOffset;
			--remaining;
		}
    
    }

   // applySymbolRange(EOF_CHAR, r,lower,upper);    /* encode an EOF */
   // writeEncodedBits(&bfpOut,lower,upper,underflowBits);

    writeRemaining(&bfpOut,lower,upper,underflowBits);         /* write out least significant bits */
    writeClose(&bfpOut);
	length = bfpOut.fp-outFile;

	
	write(length, outFile ,2);
	write(size, outFile +2,2);

	return length;
}



__host__ __device__ __inline int getChar(BitPointer *stream){
	//;
	
	int x = stream->fp[0];
		++stream->fp;

	return x;

}


/***************************************************************************
*   Function   : readBit
*   Description: This function returns the next bit from the file passed as
*                a parameter.  The bit value returned is the msb in the
*                bit buffer.
*   Parameters : stream - pointer to bit stream to read from
*   Effects    : Reads next bit from bit buffer.  If the buffer is empty,
*                a new byte will be read from the file.
*   Returned   : 0 if bit == 0, 1 if bit == 1, and EOF if operation fails.
***************************************************************************/
__host__ __device__ int readBit(BitPointer *stream)
{
    int returnValue;

    if (stream->bitCount == 0)
    {
            returnValue = getChar(stream);
            stream->bitCount = 8;
            stream->bitBuffer = returnValue;
    }

    /* bit to return is msb in buffer */
    stream->bitCount--;
    returnValue = (stream->bitBuffer) >> (stream->bitCount);

    return (returnValue & 0x01);
}

/****************************************************************************
*   Function   : initializeDecoder
*   Description: This function starts the upper and lower ranges at their
*                max/min values and reads in the most significant encoded
*                bits.
*   Parameters : bfpIn - stream to read from
*   Effects    : upper, lower, and code are initialized.  The probability
*                range list will also be initialized if an adaptive model
*                will be used.
*   Returned   : TRUE for success, otherwise FALSE
****************************************************************************/
__host__ __device__  void initializeDecoder(BitPointer *bfpIn, probability_t &lower,probability_t &upper,probability_t &code  )
{
    int i;

    code = 0;

    /* read PERCISION MSBs of code one bit at a time */
    for (i = 0; i < PRECISION; i++)
    {
        code <<= 1;

        /* treat EOF like 0 */
        if(readBit(bfpIn) == 1)
        {
            code |= 1;
        }
    }

    /* start with full probability range [0%, 100%) */
    lower = 0;
    upper = ~0;         /* all ones */
}


/***************************************************************************
*   Function   : readChar
*   Description: This function returns the next byte from the file passed as
*                a parameter.
*   Parameters : stream - pointer to bit stream to read from
*   Effects    : Reads next byte from file and updates buffer accordingly.
*   Returned   : EOF if a whole byte cannot be obtained.  Otherwise,
*                the character read.
***************************************************************************/
__host__ __device__ int readChar(BitPointer *stream)
{
    int returnValue;
    unsigned char tmp;

    returnValue = getChar(stream);

    if (stream->bitCount == 0)
    {
        /* we can just get byte from file */
        return returnValue;
    }

    /* we have some buffered bits to return too */

        /* figure out what to return */
        tmp = ((unsigned char)returnValue) >> (stream->bitCount);
        tmp |= ((stream->bitBuffer) << (8 - (stream->bitCount)));

        /* put remaining in buffer. count shouldn't change. */
        stream->bitBuffer = returnValue;

        returnValue = tmp;

    return returnValue;
}

/***************************************************************************
*   Function   : readBits  (Little Endian)
*   Description: This function reads the specified number of bits from the
*                file passed as a parameter and writes them to the
*                requested memory location (LSB to MSB).
*   Parameters : stream - pointer to bit stream to read from
*                bits - address to store bits read
*                count - number of bits to read
*   Effects    : Reads bits from the bit buffer and file stream.  The bit
*                buffer will be modified as necessary.  bits is treated as
*                a little endian integer of length >= (count/8) + 1.
*   Returned   : EOF for failure, otherwise the number of bits read.  If
*                an EOF is reached before all the bits are read, bits
*                will contain every bit through the last successful read.
***************************************************************************/
__host__ __device__ int readBits(BitPointer *stream, void *bits, const unsigned int count)
{
    unsigned char *bytes/*, shifts*/;
    int offset, remaining, returnValue;

    bytes = (unsigned char *)bits;

    offset = 0;
    remaining = count;

    /* read whole bytes */
    while (remaining >= 8)
    {
        returnValue = readChar(stream);

        bytes[offset] = (unsigned char)returnValue;
        remaining -= 8;
        offset++;
    }

    if (remaining != 0)
    {
        /* read remaining bits */
        //shifts = 8 - remaining;

        while (remaining > 0)
        {
            returnValue = readBit(stream);

            bytes[offset] <<= 1;
            bytes[offset] |= (returnValue & 0x01);
            remaining--;
        }

    }

    return count;
}



/****************************************************************************
*   Function   : getUnscaledCode
*   Description: This function undoes the scaling that ApplySymbolRange
*                performed before bits were shifted out.  The value returned
*                is the probability of the encoded symbol.
*   Parameters : None
*   Effects    : None
*   Returned   : The probability of the current symbol
****************************************************************************/
__host__ __device__ probability_t getUnscaledCode(probability_t &lower,probability_t &upper,probability_t &code, probability_t& cumulativeProb)
{
     unsigned int range;        /* must be able to hold max upper + 1 */
    unsigned int unscaled;

    range = (unsigned int)(upper - lower) + 1;

    /* reverse the scaling operations from ApplySymbolRange */
    unscaled = (unsigned int)(code - lower) + 1;
    unscaled = unscaled * (unsigned int)cumulativeProb - 1;
    unscaled /= range;

    return ((probability_t)unscaled);
}

/****************************************************************************
*   Function   : getSymbolFromProbability
*   Description: Given a probability, this function will return the symbol
*                whose range includes that probability.  Symbol is found
*                binary search on probability ranges.
*   Parameters : probability - probability of symbol.
*   Effects    : None
*   Returned   : -1 for failure, otherwise encoded symbol
****************************************************************************/
__host__ __device__ __inline int getSymbolFromProbability(probability_t probability,AdaptiveProbabilityRange& r)
{
    int first, last, middle;    /* indicies for binary search */

    first = 0;
    last = UPPER(EOF_CHAR);
    middle = last >>1;

    /* binary search */
    while (last >= first)
    {
        if (probability < getRange(LOWER(middle),r))
        {
            /* lower bound is higher than probability */
            last = middle - 1;
            middle = first + ((last - first) >>1);
            continue;
        }

        if (probability >= getRange(UPPER(middle),r))
        {
            /* upper bound is lower than probability */
            first = middle + 1;
            middle = first + ((last - first) >>1);
            continue;
        }

        /* we must have found the right value */
        return middle;
    }
	
    #if _DEBUG
    assert("Unknown Symbol");
    //cuPrintf("Unknown Symbol: %d (max: %d)\n", probability, r.ranges[UPPER(EOF_CHAR)]);
    #endif
    return -1;
}


/***************************************************************************
*   Function   : readEncodedBits
*   Description: This function attempts to shift out as many code bits as
*                possible, as bits are shifted out the coded input is
*                populated with bits from the encoded file.  Only bits
*                that will be unchanged when additional symbols are decoded
*                may be shifted out.
*
*                If the n most significant bits of the lower and upper range
*                bounds match, they will not be changed when additional
*                symbols are decoded, so they may be shifted out.
*
*                Adjustments are also made to prevent possible underflows
*                that occur when the upper and lower ranges are so close
*                that decoding another symbol won't change their values.
*   Parameters : bfpOut - pointer to open binary stream to read from.
*   Effects    : The upper and lower code bounds are adjusted so that they
*                only contain only bits that will be affected by the
*                addition of a new symbol.  Replacements are read from the
*                encoded stream.
*   Returned   : None
***************************************************************************/
__host__ __device__ void readEncodedBits(BitPointer *bfpIn,probability_t &lower,probability_t &upper,probability_t &code)
{
    int nextBit;        /* next bit from encoded input */

    for (;;)
    {
        if (( upper & MASK_BIT(0)) == (lower & MASK_BIT(0)))
        {
                        /* MSBs match, allow them to be shifted out*/
        }
        else if ((lower & MASK_BIT(1)) && !(upper & MASK_BIT(1)))
        {
            /***************************************************************
            * Possible underflow condition: neither MSBs nor second MSBs
            * match.  It must be the case that lower and upper have MSBs of
            * 01 and 10.  Remove 2nd MSB from lower and upper.
            ***************************************************************/
                        lower   &= ~(MASK_BIT(0) | MASK_BIT(1));
            upper  |= MASK_BIT(1);
            code ^= MASK_BIT(1);

            /* the shifts below make the rest of the bit removal work */
        }
        else
        {
            /* nothing to shift out */
            return;
        }

        /*******************************************************************
        * Shift out old MSB and shift in new LSB.  Remember that lower has
        * all 0s beyond it's end and upper has all 1s beyond it's end.
        *******************************************************************/
        lower <<= 1;
        upper <<= 1;
        upper |= 1;
        code <<= 1;

        if ((nextBit = readBit(bfpIn)) == EOF)
        {
            /* either all bits are shifted out or error occurred */
        }
        else
        {
            code |= nextBit;                /* add next encoded bit to code */
        }
    }

    //return;
}

/***************************************************************************
*   Function   : arDecompress
*   Description: This routine opens an arithmetically encoded file, reads
*                it's header, and builds a list of probability ranges which
*                it then uses to decode the rest of the file.
*   Parameters : inFile - Pointer to stream to decode
*                outFile - Pointer to stream to write decoded output to
*   Effects    : Encoded file is decoded
*   Returned   : TRUE for success, otherwise FALSE.
***************************************************************************/
__host__ __device__ size_t arDecompress(const unsigned char *fpIn,const size_t inSize ,unsigned char *fpOut,AdaptiveProbabilityRange& r, probability_t& cumulativeProb)
{
    int c;
    probability_t unscaled;
     BitPointer bfpIn = createBitPointer((unsigned char*)fpIn+PACKET_HEADER_LENGTH);
    

		    /* initialize coder start with full probability range [0%, 100%) */
    probability_t lower;
    probability_t upper;                     /* all ones */
    probability_t code;
	unsigned char* dstPointer = fpOut;
	const size_t decompressedSize = getUncompressedSize(fpIn);

	//bfpIn->fp = ;


    /* read start of code and initialize bounds, and adaptive ranges */
   initializeDecoder(&bfpIn,lower,upper,code);

    /* decode one symbol at a time */
    while( (dstPointer-fpOut)< decompressedSize)
    {
        //printf("%02X\t%d\t%d\n", lower, upper, code);
		
		/* get the unscaled probability of the current symbol */
        unscaled = getUnscaledCode(lower,upper,code,cumulativeProb);

        /* figure out which symbol has the above probability */
        if((c = getSymbolFromProbability(unscaled,r)) == -1)
        {
            /* error: unknown symbol */
            break;
        }
		
		dstPointer[0]=c;
		++dstPointer;


        //fputc((char)c, fpOut);

        /* factor out symbol */
        applySymbolRange(c,r,lower,upper, cumulativeProb);
        readEncodedBits(&bfpIn,lower,upper,code);
    }

    return dstPointer-fpOut;
}

__global__ void garCompress(const unsigned char *source, size_t size, unsigned char *destination)
{
  
  __shared__ AdaptiveProbabilityRange sharedMemory[NUM_THREADS];
  const int index = blockIdx.x * blockDim.x+ threadIdx.x;
  const unsigned int startPosition = index*UNCOMPRESSED_PACKET_SIZE;
  AdaptiveProbabilityRange* start =  sharedMemory + threadIdx.x;
  probability_t cumProb = INITIALIZED_CUMULATIVE_PROB;
  
  if(startPosition<size){
	size_t packetSize = size-startPosition;
	  
	start[0]= INITIALIZED_RANGE[0];
	if(packetSize>UNCOMPRESSED_PACKET_SIZE){
	    packetSize = UNCOMPRESSED_PACKET_SIZE;
	}
	arCompress(source+startPosition, packetSize, destination+(index*COMPRESSED_PACKET_SIZE),*start, cumProb);
	}
}

__global__ void garDecompress(const unsigned char *source, size_t size, unsigned char *destination)
{
  
  __shared__ AdaptiveProbabilityRange sharedMemory[NUM_THREADS];
  const int index = blockIdx.x * blockDim.x+ threadIdx.x;
  const size_t startPosition = index*COMPRESSED_PACKET_SIZE;
  const unsigned char * startSrc = source+startPosition;
  AdaptiveProbabilityRange* start =  sharedMemory + threadIdx.x;
  unsigned char* data=   destination+(index*UNCOMPRESSED_PACKET_SIZE);
  probability_t cumProb = INITIALIZED_CUMULATIVE_PROB;
  if(startPosition<size){
	//size_t packetSize = size-startPosition;
	start[0]= INITIALIZED_RANGE[0];
    /*size_t packetSize = */
    arDecompress(startSrc, getCompressedSize(startSrc),data,*start,cumProb);
	 //write(packetSize, data +2,2);
	
	}
	
}

void garCompressExecutor(const unsigned char *source, size_t size, unsigned char *destination,unsigned int numBlocks){
        
        garCompress<<<numBlocks,NUM_THREADS>>>(source, size, destination);

        #ifdef _DEBUG
        getLastCudaError("Execute garCompress kenenl failed");
        #endif
}

void garDecompressExecutor(const unsigned char *source, size_t size, unsigned char *destination,unsigned int numBlocks){
        
        garDecompress<<<numBlocks,NUM_THREADS>>>(source, size, destination);

        #ifdef _DEBUG
        getLastCudaError("Execute garCompress kenenl failed");
        #endif
}
